#include "fast_pcl/ndt_gpu/MatrixDevice.h"
#include "fast_pcl/ndt_gpu/debug.h"

namespace gpu {
MatrixDevice::MatrixDevice(int rows, int cols) {
	rows_ = rows;
	cols_ = cols;
	offset_ = 1;
	fr_ = true;

	checkCudaErrors(hipMalloc(&buffer_, sizeof(double) * rows_ * cols_ * offset_));
	checkCudaErrors(hipMemset(buffer_, 0, sizeof(double) * rows_ * cols_ * offset_));
	checkCudaErrors(hipDeviceSynchronize());
}


void MatrixDevice::memFree()
{
	if (fr_) {
		if (buffer_ != NULL)
			checkCudaErrors(hipFree(buffer_));
	}
}


SquareMatrixDevice::SquareMatrixDevice(int size) :
	MatrixDevice(size, size)
{

}

}
